#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <stdlib.h>  

#define MOD 1000000007   // defining the modulus value for the MST cost output
#define BLOCK_SIZE 1024     // block size for kernel launches
using namespace std;

struct Edge {
    int src, dest, weight;
};

__device__ int Swap_Function(int* address, int val) { // This swap function (using atomicCAS) exchanges the value at the given address with 'val' and returns the old value.
    int oldAddress = *address;
    int assumed;
    do {
        assumed = oldAddress;
        oldAddress = atomicCAS(address, assumed, val);
    } while (assumed != oldAddress);
    return oldAddress;
}

__device__ bool Union_Operation(int* parent, int Node1, int Node2) {   // performs the union of two disjoint sets. Returns true if the union was successful, false if both nodes were already in the same set.
    for(;;) {     // loop until an 'union' operation is performed or the nodes are found to be in the same set
        for(;;) {    // performing path compression for Node1
            int p = parent[Node1];
            if (p == Node1) {
                break;
            }
            int newp = parent[p];
            Swap_Function(&parent[Node1], newp);
            Node1 = newp;
        }
    
        for(;;) {     // performing path compression for Node2
            int p = parent[Node2];
            if (p == Node2) {
                break;
            }
            int newp = parent[p];
            Swap_Function(&parent[Node2], newp);
            Node2 = newp;
        }
        
        if (Node1 == Node2) {  
            return false;
        }

        int high = (Node1 > Node2) ? Node1 : Node2;
        int low = (Node1 < Node2) ? Node1 : Node2;
        int old = atomicCAS(&parent[high], high, low);  // link the higher-indexed node to the lower-indexed node.
        if (old == high){
            return true;
        }
        // if another thread have updated the parent, then repeat the process
    }
}

__device__ int adjust_weight(long long base, int factor) {
    int mod_result = (int)((base * factor) % MOD);
    return mod_result;
}

__global__ void EdgeWeight(int* deviceEdges, int* deviceType, int E) {   // kernel to adjust the weight of each edge (based on the terrain type) in parallel
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < E) 
    {
        int index = tid * 3;    // starting index
        int baseWeight = deviceEdges[index + 2];
        long long longBaseweight = baseWeight * 1LL;   // long long to avoid overflow during multiplication operation

        // adjusting weight based on terrain type below
        if (deviceType[tid] == 1) {        // terrain is green
            deviceEdges[index + 2] = adjust_weight(longBaseweight, 2);
        } 
        else if (deviceType[tid] == 2) {   // terrain is traffic
            deviceEdges[index + 2] = adjust_weight(longBaseweight, 5);
        }
        else if (deviceType[tid] == 3) {   // terrain is dept
            deviceEdges[index + 2] = adjust_weight(longBaseweight, 3);
        } 
        else {                             // terrain is normal
            deviceEdges[index + 2] = adjust_weight(longBaseweight, 1);
        }
    }
}

__global__ void precompute(int* deviceParent, unsigned long long* lowestValue, int V, int choice) {     // kernel to initialize two arrays based on the choice (either 0 or 1)
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < V) {
        if (choice == 0)
            deviceParent[tid] = tid;     // if choice is 0, the deviceParent array is initialized ina way such that each vertex is its own parent
        else
            lowestValue[tid] = 0xFFFFFFFFFFFFFFFFULL;    // if choice is 1, the lowestValue array for each vertex is initialized to the maximum value (i.e., 0xFFFFFFFFFFFFFFFFULL)
    }
}


__global__ void CheapestOutgoingEdge(int* deviceEdges, int E, int* deviceParent, unsigned long long* lowestValue) {    // kernel to scan each edge and update the lowestValue array (using atomicMin) for the components of its endpoints(if the endpoints belong to different sets)
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < E) {
        int index = tid * 3;     // starting index for this edge
        int source = deviceEdges[index];           // Source vertex
        int destination = deviceEdges[index + 1];  // Destination vertex
        int weight = deviceEdges[index + 2];       // Weight of the edge

        int Updated_Source;
        for(;;) {          // path compression for the source vertex
            int p = deviceParent[source];
            if (p == source) {
                Updated_Source = source;
                break;
            }
            int newp = deviceParent[p];
            Swap_Function(&deviceParent[source], newp);
            source = newp;
        }
       
        int Updated_Destination;
        for(;;) {      // path compression for destination vertex
            int p = deviceParent[destination];
            if (p == destination) {
                Updated_Destination = destination;
                break;
            }
            int newp = deviceParent[p];
            Swap_Function(&deviceParent[destination], newp);
            destination = newp;
        }

        if (Updated_Source != Updated_Destination) {
            unsigned long long newWeight = ((unsigned long long) weight) << 32;
            unsigned long long res = ((unsigned int) tid) | newWeight;

            // updating the lowest value for both source and destination 
            atomicMin(&lowestValue[Updated_Source], res);
            atomicMin(&lowestValue[Updated_Destination], res);
        }
    }
}

__global__ void setMinimumCostEdge(int* deviceEdges, int* deviceParent, unsigned long long* lowestValue, int V, int* numTrees, int* totalCost) {     // Kernel to apply the cheapest edge for each vertex. If a candidate is not INF in lowestValue array, it will extract the edge index and retrieve the edge details
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < V) {
        unsigned long long candidate = lowestValue[tid];
        if (candidate != 0xFFFFFFFFFFFFFFFFULL) {
            int edgeIndex = (int)(0xFFFFFFFF & candidate);
            int edgeindex = edgeIndex * 3;            // starting index
            int source = deviceEdges[edgeindex];           // Source vertex
            int destination = deviceEdges[edgeindex + 1];  // Destination vertex
            int weight = deviceEdges[edgeindex + 2];       // Edge weight

            int Updated_Source;
            for(;;) {           // path compression for the source vertex
                int p = deviceParent[source];
                if (p == source) {
                    Updated_Source = source;
                    break;
                }
                int newp = deviceParent[p];
                Swap_Function(&deviceParent[source], newp);
                source = newp;
            }
            
            int Updated_Destination;
            for(;;) {           //path compression for the destination vertex
                int p = deviceParent[destination];
                if (p == destination) {
                    Updated_Destination = destination;
                    break;
                }
                int newp = deviceParent[p];
                Swap_Function(&deviceParent[destination], newp);
                destination = newp;
            }
            
            if (Updated_Source != Updated_Destination) {       // if the endpoints belong to different components, perform union operatoin
                if (Union_Operation(deviceParent, Updated_Source, Updated_Destination)) {
                    atomicAdd(totalCost, weight);   // if union successful, then add the edge's weight to total cost
                    atomicAdd(numTrees, -1);        // if union successful, then decrement the number of components
                }
            }
        }
    }
}

void kernelLaunches(int* deviceEdges, int E, int* deviceParent, unsigned long long* lowestValue, int V, int* numTrees, int* totalCost, int numBlocksV, int blocksE) {
    precompute<<<numBlocksV, BLOCK_SIZE>>>(deviceParent, lowestValue, V, 1);    // kernel launch to reinitialize the 'lowestvalue' parent array based on the choice value
    CheapestOutgoingEdge<<<blocksE, BLOCK_SIZE>>>(deviceEdges, E, deviceParent, lowestValue);    // kernel launch to find the minimum cost outgoing edge for each component
    setMinimumCostEdge<<<numBlocksV, BLOCK_SIZE>>>(deviceEdges, deviceParent, lowestValue, V, numTrees, totalCost);     // kernel launch to apply the selected edges to merge components and update the MST total cost
}

int main() {
    int V, E;
    cin >> V >> E;
    
    int* hostEdges = (int*)malloc(E * 3 * sizeof(int));  // each edge is represented by 3 integers (source, destination, weight)
    int* types = (int*)malloc(E * sizeof(int));
    
    // taking input below (edges and terrain type for each edge)
    for (int i = 0; i < E; i++) {
        int u, v, wt;
        string terrainType;
        cin >> u >> v >> wt >> terrainType;
        int type = (terrainType == "green") ? 1 : ((terrainType == "traffic") ? 2 : ((terrainType == "dept") ? 3 : 0));  // Terrain types considered: 0 for normal, 1 for green, 2 for traffic and 3 for dept
        types[i] = type;
        //storing edge information in the 'hostEdges' array.
        hostEdges[i * 3 + 0] = u;
        hostEdges[i * 3 + 1] = v;
        hostEdges[i * 3 + 2] = wt;
    }
    
    // allocating memory for device (edges, terrain types, parent array, lowest edge-value array, and counters for number of trees and total cost)
    int* deviceEdges;
    int* deviceType;
    int* deviceParent;
    unsigned long long* lowestValue;
    int* numTrees;
    int* totalCost;
    
    hipMalloc(&deviceEdges, E * 3 * sizeof(int));
    hipMalloc(&deviceType, E * sizeof(int));
    hipMalloc(&deviceParent, V * sizeof(int));
    hipMalloc(&lowestValue, V * sizeof(unsigned long long));
    hipMalloc(&numTrees, sizeof(int));
    hipMalloc(&totalCost, sizeof(int));
    
    // calculating grid dimensions for the vertices and edges
    int numBlocksV = ceil((float)V / BLOCK_SIZE);
    int blocksE = ceil((float)E / BLOCK_SIZE);
    
    // copying data from host to device 
    hipMemcpy(deviceEdges, hostEdges, E * 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceType, types, E * sizeof(int), hipMemcpyHostToDevice);
    
    int host_numTrees = V;
    int host_Actual_Cost = 0;
    hipMemcpy(numTrees, &host_numTrees, sizeof(int), hipMemcpyHostToDevice);
    hipMemset(totalCost, 0, sizeof(int));
    
    auto start = std::chrono::high_resolution_clock::now();
    
    EdgeWeight<<<blocksE, BLOCK_SIZE>>>(deviceEdges, deviceType, E);          // kernel launch to adjust edge weights based on terrain types
    precompute<<<numBlocksV, BLOCK_SIZE>>>(deviceParent, lowestValue, V, 0);  // kernel launch to initialize the 'lowestvalue' parent array, since choice is 0, so each vertex will be its own parent
    
    while(host_numTrees > 1) {  // loop until all vertices are merged into a single component
        kernelLaunches(deviceEdges, E, deviceParent, lowestValue, V, numTrees, totalCost, numBlocksV, blocksE);
        hipMemcpy(&host_numTrees, numTrees, sizeof(int), hipMemcpyDeviceToHost);
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    hipMemcpy(&host_Actual_Cost, totalCost, sizeof(int), hipMemcpyDeviceToHost);   // copying back the final MST cost from device to host
    
    cout << host_Actual_Cost % MOD << endl;
    //cout << "Execution Time: " << elapsed.count() << "s\n";
    
    // free all the allocated device and host memory
    hipFree(deviceEdges);
    hipFree(deviceType);
    hipFree(deviceParent);
    hipFree(lowestValue);
    hipFree(numTrees);
    hipFree(totalCost);
    free(hostEdges);
    free(types);
    
    return 0;
}
